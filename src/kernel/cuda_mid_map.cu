#include "hip/hip_runtime.h"
#include <ubfs/cuda_mid_map.cuh>

NF1Map3D::NF1Map3D(int3 m_map_size_, float grid_step_):
  m_map_size(m_map_size_), grid_step(grid_step_)
{
  m_volume =m_map_size.x*m_map_size.y*m_map_size.z;
  m_byte_size = m_volume*static_cast<int>(sizeof(float));
  m_color_size = m_volume*static_cast<int>(sizeof(int));
  m_flg_size =m_volume*static_cast<int>(sizeof(bool));
  m_edt_size = m_volume*static_cast<int>(sizeof(SeenDist));
  m_max_width = m_map_size.x+m_map_size.y+m_map_size.z;


}

void NF1Map3D::setDefaut()
{
  CUDA_DEV_MEMSET(d_cost_to_go,FLT_MAX,static_cast<size_t>(m_byte_size));
  CUDA_DEV_MEMSET(d_color,WHITE,static_cast<size_t>(m_color_size));
  CUDA_DEV_MEMSET(d_obsflg,0,static_cast<size_t>(m_flg_size));
  //    CUDA_DEV_MEMSET(d_val_map,0,static_cast<size_t>(m_edt_size));
    thrust::fill(obs_vec.begin(),obs_vec.end(),EMPTY_KEY);// can be deleted!!!!!
    thrust::fill(obs_vec_dense.begin(),obs_vec_dense.end(),EMPTY_KEY);

}
void NF1Map3D::setup_device()
{
  CUDA_ALLOC_DEV_MEM(&d_cost_to_go,static_cast<size_t>(m_byte_size));
  CUDA_DEV_MEMSET(d_cost_to_go,FLT_MAX,static_cast<size_t>(m_byte_size));

  CUDA_ALLOC_DEV_MEM(&d_color,static_cast<size_t>(m_color_size));
  CUDA_DEV_MEMSET(d_color,WHITE,static_cast<size_t>(m_color_size));

  CUDA_ALLOC_DEV_MEM(&d_obsflg,static_cast<size_t>(m_flg_size));
  CUDA_DEV_MEMSET(d_obsflg,0,static_cast<size_t>(m_flg_size));

  obs_vec.resize(m_volume);
  thrust::fill(obs_vec.begin(),obs_vec.end(),EMPTY_KEY);

  obs_vec_dense.resize(m_volume);
  thrust::fill(obs_vec_dense.begin(),obs_vec_dense.end(),EMPTY_KEY);

  obs_dense_h.resize(m_volume);

  this->d_obs =thrust::raw_pointer_cast(&obs_vec[0]);
  this->d_obs_dense =thrust::raw_pointer_cast(&obs_vec_dense[0]);

  CUDA_ALLOC_DEV_MEM(&d_val_map,static_cast<size_t>(m_edt_size));
  CUDA_DEV_MEMSET(d_val_map,0,static_cast<size_t>(m_edt_size));

  CUDA_ALLOC_DEV_MEM(&d_dirs_3d,num_dirs_3d*sizeof(int3));
  CUDA_MEMCPY_H2D(d_dirs_3d,dirs_3d,num_dirs_3d*sizeof(int3));

  //  hipHostAlloc((**void)&h_cost_to_go,m_byte_size,hipHostMallocDefault);
    h_cost_to_go = new float[m_byte_size];
}

void NF1Map3D::free_device()
{
  CUDA_FREE_DEV_MEM(d_cost_to_go);
  CUDA_FREE_DEV_MEM(d_color);
  CUDA_FREE_DEV_MEM(d_obsflg);
  CUDA_FREE_DEV_MEM(d_val_map);

  CUDA_FREE_DEV_MEM(d_dirs_3d);

  delete [] h_cost_to_go;

}



