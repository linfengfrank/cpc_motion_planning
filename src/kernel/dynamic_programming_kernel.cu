#include "hip/hip_runtime.h"
#include <cpc_motion_planning/dynamic_programming.cuh>
#define DT 0.05
namespace GPU_DP
{
template <int N>
__global__
void test(VoidPtrCarrier<N> data)
{
  CUDA_MAT::Mat4Act *S_A = static_cast<CUDA_MAT::Mat4Act*>(data[0]);
  CUDA_MAT::Mat4f *S_old = static_cast<CUDA_MAT::Mat4f*>(data[1]);
  CUDA_MAT::Mat4f *S_new = static_cast<CUDA_MAT::Mat4f*>(data[2]);
  CUDA_MAT::Vecf *bin_p = static_cast<CUDA_MAT::Vecf*>(data[3]);
  CUDA_MAT::Vecf *bin_v = static_cast<CUDA_MAT::Vecf*>(data[4]);
  CUDA_MAT::Vecf *bin_theta = static_cast<CUDA_MAT::Vecf*>(data[5]);
  CUDA_MAT::Vecf *bin_w = static_cast<CUDA_MAT::Vecf*>(data[6]);

  float s_curr[4];
  s_curr[0] =  pos_gen_val(blockIdx.x);
  s_curr[1] =  vel_gen_val(blockIdx.y);
  s_curr[2] =  theta_gen_val(blockIdx.z);
  s_curr[3] =  w_gen_val(threadIdx.x);


  float val;
  float s_next[4];
  float val_min = 1e6;
  float acc_lat;
  float acc_tot;
  dp_action best_action;
  bool updated = false;

  for (float acc=-2;acc<2.1;acc+=0.2)
  {
    for (float alpha=-3;alpha<3.1;alpha+=0.3)
    {
      s_next[0] = s_curr[0] + s_curr[1]*DT + 0.5*acc*DT*DT;
      s_next[1] = s_curr[1] + acc*DT;
      s_next[2] = s_curr[2] + s_curr[3]*DT + 0.5*alpha*DT*DT;
      s_next[3] = s_curr[3] + alpha*DT;
      val = CUDA_MAT::get_value(s_next,*S_old, *bin_p, *bin_v, *bin_theta, *bin_w);
      val += 10*acc*acc + 10*alpha*alpha;
      val += 1*s_curr[0]*s_curr[0] + 0.2*s_curr[1]*s_curr[1] +s_curr[2]*s_curr[2] + 0.2*s_curr[3]*s_curr[3];
      if (s_curr[1] - 4.0 > 0)
        val += 80*(s_curr[1] - 4.0);

      if (s_curr[1] < -4.0)
        val += 80*(-s_curr[1] - 4.0);

      if (s_curr[3] - 2 > 0)
        val += 80*(s_curr[3] - 2);

      if (s_curr[3] < -2)
        val += 80*(-s_curr[3] - 2);

      acc_lat = s_curr[1]*s_curr[3];
      acc_tot = sqrt(acc_lat*acc_lat + acc*acc);

//      if (fabs(acc_lat) > 2)
//        val += 80*(fabs(acc_lat) - 2);

      if (acc_tot - 1.5 > 0)
        val += 80*(acc_tot - 1.5);

//      val+= 0.5*acc_tot*acc_tot;

      if (fabs(s_curr[0]) > 0.25 || fabs(s_curr[1]) > 0.25 || fabs(s_curr[2]) > 0.25 || fabs(s_curr[3]) > 0.25)
      {
        val += 32;
      }

      if (val < val_min)
      {
        updated = true;
        val_min = val;
        best_action.acc = acc;
        best_action.alpha = alpha;
      }
    }
  }

  CUDA_MAT::mat4f_get_val(blockIdx.x,blockIdx.y,blockIdx.z,threadIdx.x,*S_new) = val_min;

  if (updated)
    CUDA_MAT::mat4act_get_val(blockIdx.x,blockIdx.y,blockIdx.z,threadIdx.x,*S_A) = best_action;

  //printf("%f\n",val);
}

template<int N>
void program(VoidPtrCarrier<N> ptr_car, size_t *bin_size)
{
  dim3 grid_size;
  grid_size.x = bin_size[0];
  grid_size.y = bin_size[1];
  grid_size.z = bin_size[2];

  dim3 block_size;
  block_size.x = bin_size[3];
  block_size.y = 1;
  block_size.z = 1;



  for (int i=0; i<140; i++)
  {
    printf("Iteration %d\n",i);
    if (i % 2 == 0)
    {
      void* tmp = ptr_car[2];
      ptr_car[2] = ptr_car[1];
      ptr_car[1] = tmp;
      test<N><<<grid_size,block_size>>>(ptr_car);
    }
    else
    {
      void* tmp = ptr_car[2];
      ptr_car[2] = ptr_car[1];
      ptr_car[1] = tmp;
      test<N><<<grid_size,block_size>>>(ptr_car);
    }

    hipDeviceSynchronize();
  }
}
}
template void GPU_DP::program<7>(VoidPtrCarrier<7> ptr_car, size_t *bin_size);

