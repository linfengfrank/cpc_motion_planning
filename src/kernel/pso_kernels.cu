#include "hip/hip_runtime.h"
#include <cpc_motion_planning/pso/pso_kernels.cuh>
#include <cuda_geometry/hip/hip_vector_types.h>
#include <cpc_motion_planning/uav/uav_single_target_evluator.h>
namespace PSO
{
//---
template<class Model, class Controler, class Evaluator, class TmpSwarm>
__host__ __device__
float evaluate_trajectory(const EDTMap &map, const Evaluator &eva, Model &m, const Controler &ctrl, const typename TmpSwarm::Trace &ttr)
{
  typename Model::State s = m.get_ini_state();
  float cost = 0;
  float dt = PSO_SIM_DT;
  //float3 goal_p = goal.s.p;
  for (float t=0.0f; t<PSO_TOTAL_T; t+=dt)
  {
    int i = static_cast<int>(floor(t/PSO_STEP_DT));
    if (i > PSO_STEPS - 1)
      i = PSO_STEPS - 1;

    float3 u = ctrl.dp_control(s, ttr[i]);
    m.model_forward(s,u,dt);

    cost += 0.1*sqrt(u.x*u.x + u.y*u.y + u.z*u.z);
    cost += eva.process_cost(s,map);

//    float3 ctr_pnt = tr[i];
//    float3 diff_tr = ctr_pnt - goal_p;

//    cost+= 0.05*sqrt(diff_tr.x*diff_tr.x + diff_tr.y*diff_tr.y + diff_tr.z*diff_tr.z);
  }
  cost += eva.final_cost(s,map);
//  Trace diff = tr - last_tr;
//  cost += sqrt(diff.square())/static_cast<float>(PSO_STEPS);
  return cost;
}

//---
template<class TmpSwarm>
__global__
void setup_random_states_kernel(typename TmpSwarm::Particle* tptcls)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(9876, idx, 0, &(tptcls[idx].rs));
}

//---
template<class Model, class Controler, class Evaluator, class TmpSwarm>
__global__
void initialize_particles_kernel(bool first_run,
                                 EDTMap map, Evaluator eva, Model m, Controler ctrl, TmpSwarm tsw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (first_run || idx != tsw.ptcl_size-1)
  {
    tsw.initialize_a_particle(m.get_ini_state(),tsw.ptcls[idx]);
  }
  tsw.ptcls[idx].best_cost = evaluate_trajectory<Model,Controler,Evaluator,TmpSwarm>(map,eva,m, ctrl, tsw.ptcls[idx].best_loc);
}

//---
template<class Model, class Controler, class Evaluator, class TmpSwarm>
__global__
void iterate_particles_kernel(float weight,
                              EDTMap map, Evaluator eva, Model m, Controler ctrl, TmpSwarm tsw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (idx == tsw.ptcl_size-1)
    return;

  float r1 = tsw.rand_float_gen(&(tsw.ptcls[idx].rs),0,1);
  float r2 = tsw.rand_float_gen(&(tsw.ptcls[idx].rs),0,1);

  tsw.ptcls[idx].ptcl_vel =
      tsw.ptcls[idx].ptcl_vel*weight -
      (tsw.ptcls[idx].curr_loc - tsw.ptcls[idx].best_loc)*r1 -
      (tsw.ptcls[idx].curr_loc - tsw.ptcls[tsw.ptcl_size-1].curr_loc)*r2;

  tsw.bound_ptcl_velocity(tsw.ptcls[idx]);

  tsw.ptcls[idx].curr_loc = tsw.ptcls[idx].curr_loc + tsw.ptcls[idx].ptcl_vel;

  tsw.bound_ptcl_location(m.get_ini_state(), tsw.ptcls[idx]);

  float cost = evaluate_trajectory<Model,Controler,Evaluator,TmpSwarm>(map,eva,m,ctrl,tsw.ptcls[idx].curr_loc);

  if (cost < tsw.ptcls[idx].best_cost)
  {
    tsw.ptcls[idx].best_cost = cost;
    tsw.ptcls[idx].best_loc = tsw.ptcls[idx].curr_loc;
  }
}

//---------
template<class TmpSwarm>
__global__
void copy_best_value_kernel(float* best_values, TmpSwarm tsw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  best_values[idx] = tsw.ptcls[idx].best_cost;
}

//---------
template<class TmpSwarm>
void setup_random_states(const TmpSwarm &tsw)
{
  setup_random_states_kernel<TmpSwarm><<<1,tsw.ptcl_size>>>(tsw.ptcls);
}

//---------
template<class Model, class Controler, class Evaluator, class TmpSwarm>
void initialize_particles(bool first_run,
                          const EDTMap &map,const Evaluator &eva, const Model &m, const Controler &ctrl, const TmpSwarm &tsw)
{
  initialize_particles_kernel<Model,Controler,Evaluator,TmpSwarm><<<1,tsw.ptcl_size>>>(first_run,map,eva,m,ctrl,tsw);
}

//---------
template<class Model, class Controler, class Evaluator, class TmpSwarm>
void iterate_particles(float weight,
                       const EDTMap &map, const Evaluator &eva, const Model &m, const Controler &ctrl, const TmpSwarm &tsw)
{
  iterate_particles_kernel<Model,Controler,Evaluator,TmpSwarm><<<1,tsw.ptcl_size>>>(weight,map,eva,m,ctrl,tsw);
}

//---------
template<class TmpSwarm>
void copy_best_values(float *best_values, const TmpSwarm &tsw)
{
  copy_best_value_kernel<TmpSwarm><<<1,tsw.ptcl_size>>>(best_values,tsw);
}

//float evaluate_trajectory_wrapper(const UAVModel::State &s0, const Trace &tr, VoidPtrCarrier ptr_car,const UniformBinCarrier &ubc,
//               const EDTMap &map, const Trace &last_tr)
//{
//  return 0;
////  return evaluate_trajectory(s0, goal, tr, ptr_car,ubc,
////                             map, last_tr);
//}

}

template void PSO::initialize_particles<UAV::UAVModel, UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(bool first_run,
                                                                       const EDTMap &map, const UAV::SingleTargetEvaluator &eva, const UAV::UAVModel &m, const UAV::UAVDPControl &ctrl , const  UAV::UAVSwarm<1> &tsw);


template void PSO::iterate_particles<UAV::UAVModel,  UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(float weight,
                                                                    const EDTMap &map,const UAV::SingleTargetEvaluator &eva, const UAV::UAVModel &m,const UAV::UAVDPControl &ctrl , const  UAV::UAVSwarm<1> &tsw);

template float PSO::evaluate_trajectory<UAV::UAVModel,  UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(
                          const EDTMap &map, const UAV::SingleTargetEvaluator &eva, UAV::UAVModel &m,const UAV::UAVDPControl &ctrl, const UAV::UAVSwarm<1>::Trace &ttr);

template void PSO::setup_random_states< UAV::UAVSwarm<1> >(const UAV::UAVSwarm<1> &tsw);

template void PSO::copy_best_values< UAV::UAVSwarm<1> >(float *best_values, const UAV::UAVSwarm<1> &tsw);
