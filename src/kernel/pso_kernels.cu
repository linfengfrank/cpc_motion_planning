#include "hip/hip_runtime.h"
#include <cpc_motion_planning/pso/pso_kernels.cuh>
#include <cuda_geometry/hip/hip_vector_types.h>

namespace PSO
{
//---
template<class Swarm>
__global__
void setup_random_states_kernel(typename Swarm::Particle* tptcls)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(9876, idx, 0, &(tptcls[idx].rs));
}

//---
template<class Model, class Controller, class Evaluator, class Swarm>
__global__
void initialize_particles_kernel(bool first_run,
                                 EDTMap map, Evaluator eva, Model m, Controller ctrl, Swarm sw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  // Initialize the particles curr_loc, ptcl_vel and best_loc
  sw.initialize_a_particle(m.get_ini_state(),sw.ptcls[idx]);

  // Initialize the glb_best only at first run together with the first particle
  if (first_run && idx == 0)
    sw.initialize_a_particle(m.get_ini_state(),*(sw.best_ptcl));

  // Evaluate the initialized particle
  sw.ptcls[idx].best_cost = ctrl.template simulate_evaluate<Model,Evaluator,Swarm >(map,eva,m, sw, sw.ptcls[idx].best_loc, sw.ptcls[idx].collision);

  // Evaluate the best particle together with the first one
  if (idx == 0)
    sw.best_ptcl->best_cost = ctrl.template simulate_evaluate<Model,Evaluator,Swarm >(map,eva,m, sw, sw.best_ptcl->best_loc, sw.best_ptcl->collision);
}

//---
template<class Model, class Controller, class Evaluator, class Swarm>
__global__
void iterate_particles_kernel(float weight,
                              EDTMap map, Evaluator eva, Model m, Controller ctrl, Swarm sw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  float r1 = PSO::rand_float_gen(&(sw.ptcls[idx].rs),0,1);
  float r2 = PSO::rand_float_gen(&(sw.ptcls[idx].rs),0,1);

  sw.ptcls[idx].ptcl_vel =
      sw.ptcls[idx].ptcl_vel*weight -
      (sw.ptcls[idx].curr_loc - sw.ptcls[idx].best_loc)*r1 -
      (sw.ptcls[idx].curr_loc - sw.best_ptcl->best_loc)*r2;

  sw.bound_ptcl_velocity(sw.ptcls[idx]);

  sw.ptcls[idx].curr_loc = sw.ptcls[idx].curr_loc + sw.ptcls[idx].ptcl_vel;

  sw.bound_ptcl_location(m.get_ini_state(), sw.ptcls[idx]);

  float cost = ctrl.template simulate_evaluate<Model,Evaluator,Swarm>(map,eva,m,sw,sw.ptcls[idx].curr_loc,sw.ptcls[idx].collision);

  if (cost < sw.ptcls[idx].best_cost && !sw.ptcls[idx].collision)
  {
    sw.ptcls[idx].best_cost = cost;
    sw.ptcls[idx].best_loc = sw.ptcls[idx].curr_loc;
  }
}

//---
template<class Model, class Controller, class Evaluator, class Swarm>
__global__
void evaluate_ptcl_kernel(typename Swarm::Particle ptc, float* cost, EDTMap map, Evaluator eva, Model m, Controller ctrl, Swarm sw)
{
  *cost = ctrl.template simulate_evaluate<Model,Evaluator,Swarm>(map,eva,m,sw,ptc.curr_loc, ptc.collision);
}

//---------
template<class Swarm>
__global__
void copy_best_value_kernel(float* best_values, Swarm sw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  best_values[idx] = sw.ptcls[idx].best_cost;
}

//---------
template<class Swarm>
__global__
void update_glb_best_kernel(int sw_bst_id, Swarm sw)
{
  // Do not copy the random generator
  if (sw.ptcls[sw_bst_id].best_cost < sw.best_ptcl->best_cost)
  {
    sw.best_ptcl->curr_loc = sw.ptcls[sw_bst_id].curr_loc;
    sw.best_ptcl->best_loc = sw.ptcls[sw_bst_id].best_loc;
    sw.best_ptcl->ptcl_vel = sw.ptcls[sw_bst_id].ptcl_vel;
    sw.best_ptcl->best_cost = sw.ptcls[sw_bst_id].best_cost;
    sw.best_ptcl->collision = sw.ptcls[sw_bst_id].collision;
  }
}

//---------
template<class Swarm>
void setup_random_states(const Swarm &sw)
{
  setup_random_states_kernel<Swarm><<<1,sw.ptcl_size>>>(sw.ptcls);
}

//---------
template<class Model, class Controller, class Evaluator, class Swarm>
void initialize_particles(bool first_run,
                          const EDTMap &map,const Evaluator &eva, const Model &m, const Controller &ctrl, const Swarm &sw)
{
  initialize_particles_kernel<Model,Controller,Evaluator,Swarm><<<1,sw.ptcl_size>>>(first_run,map,eva,m,ctrl,sw);
}

//---------
template<class Model, class Controller, class Evaluator, class Swarm>
void iterate_particles(float weight,
                       const EDTMap &map, const Evaluator &eva, const Model &m, const Controller &ctrl, const Swarm &sw)
{
  iterate_particles_kernel<Model,Controller,Evaluator,Swarm><<<1,sw.ptcl_size>>>(weight,map,eva,m,ctrl,sw);
}

//---------
template<class Swarm>
void copy_best_values(float *best_values, const Swarm &sw)
{
  copy_best_value_kernel<Swarm><<<1,sw.ptcl_size>>>(best_values,sw);
}

template<class Swarm>
void update_glb_best(int sw_best_idx, const Swarm &sw)
{
  update_glb_best_kernel<Swarm><<<1,1>>>(sw_best_idx,sw);
}

template<class Model, class Controller, class Evaluator, class Swarm>
float evaluate_particle(const typename Swarm::Particle &ptc, const EDTMap &map, const Evaluator &eva, const Model &m, const Controller &ctrl, const Swarm &sw)
{
  float *cost_dev, cost_hst;
  CUDA_ALLOC_DEV_MEM(&cost_dev, sizeof (float));
  evaluate_ptcl_kernel<Model,Controller,Evaluator,Swarm><<<1,1>>>(ptc,cost_dev,map,eva,m,ctrl,sw);
  CUDA_MEMCPY_D2H(&cost_hst,cost_dev,sizeof(float));
  CUDA_FREE_DEV_MEM(cost_dev);
  return cost_hst;
}

}

#define INST_initialize_particles(M,C,E,S) template void PSO::initialize_particles< M,C,E,S > \
(bool, const EDTMap&, const E&, const M&, const C& , const S&);

#define INST_iterate_particles(M,C,E,S) template void PSO::iterate_particles< M,C,E,S > \
(float, const EDTMap&, const E&, const M&, const C& , const S&); \
template float PSO::evaluate_particle< M,C,E,S > \
(const typename S::Particle&, const EDTMap&, const E&, const M&, const C& , const S&);

#define INST_setup_random_states(S) template void PSO::setup_random_states< S > \
(const S&);

#define INST_copy_best_values(S) template void PSO::copy_best_values< S > \
(float *best_values, const S&); \
template void PSO::update_glb_best< S > \
(int, const S&);

#define INST_group(M,C,E,S) INST_initialize_particles(M,C,E,S) \
  INST_iterate_particles(M,C,E,S) \
  INST_setup_random_states(S) \
  INST_copy_best_values(S)

INST_group(UAV::UAVModel, UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1>);
INST_group(UAV::UAVModel, UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<2>);
INST_group(UAV::UAVModel, UAV::UAVDPVelControl, UAV::VelocityEvaluator, UAV::UAVVelSwarm<2>);

INST_initialize_particles(UAV::UAVModel, UAV::UAVJLTControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1>);
INST_iterate_particles(UAV::UAVModel, UAV::UAVJLTControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1>);

INST_initialize_particles(UAV::UAVModel, UAV::UAVJLTControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<2>);
INST_iterate_particles(UAV::UAVModel, UAV::UAVJLTControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<2>);

INST_initialize_particles(UAV::UAVModel, UAV::UAVDPControl, UAV::NF1Evaluator, UAV::UAVSwarm<1>);
INST_iterate_particles(UAV::UAVModel, UAV::UAVDPControl, UAV::NF1Evaluator, UAV::UAVSwarm<1>);

INST_initialize_particles(UAV::UAVModel, UAV::UAVJLTControl, UAV::NF1Evaluator, UAV::UAVSwarm<1>);
INST_iterate_particles(UAV::UAVModel, UAV::UAVJLTControl, UAV::NF1Evaluator, UAV::UAVSwarm<1>);

INST_initialize_particles(UAV::UAVModel, UAV::UAVDPControl, UAV::CorridorEvaluator, UAV::UAVSwarm<1>);
INST_iterate_particles(UAV::UAVModel, UAV::UAVDPControl, UAV::CorridorEvaluator, UAV::UAVSwarm<1>);
