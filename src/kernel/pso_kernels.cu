#include "hip/hip_runtime.h"
#include <cpc_motion_planning/pso/pso_kernels.cuh>
#include <cuda_geometry/hip/hip_vector_types.h>
#include <cpc_motion_planning/uav/uav_single_target_evluator.h>
namespace PSO
{
//---
template<class Model, class Controler, class Evaluator, class TmpSwarm>
__host__ __device__
float evaluate_trajectory(const Trace &tr,
                          const EDTMap &map, const Trace &last_tr, const Evaluator &eva, Model &m, const Controler &ctrl, const typename TmpSwarm::Trace &ttr)
{
  typename Model::State s = m.get_ini_state();
  float cost = 0;
  float dt = PSO_SIM_DT;
  //float3 goal_p = goal.s.p;
  for (float t=0.0f; t<PSO_TOTAL_T; t+=dt)
  {
    int i = static_cast<int>(floor(t/PSO_STEP_DT));
    if (i > PSO_STEPS - 1)
      i = PSO_STEPS - 1;

    float3 u = ctrl.dp_control(s, tr[i]);
    m.model_forward(s,u,dt);

    cost += 0.1*sqrt(u.x*u.x + u.y*u.y + u.z*u.z);
    cost += eva.process_cost(s,map);

//    float3 ctr_pnt = tr[i];
//    float3 diff_tr = ctr_pnt - goal_p;

//    cost+= 0.05*sqrt(diff_tr.x*diff_tr.x + diff_tr.y*diff_tr.y + diff_tr.z*diff_tr.z);
  }
  cost += eva.final_cost(s,map);
//  Trace diff = tr - last_tr;
//  cost += sqrt(diff.square())/static_cast<float>(PSO_STEPS);
  return cost;
}

//---
template<class TmpSwarm>
__global__
void setup_random_states_kernel(Particle *ptcls, TmpSwarm tsw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(9876, idx, 0, &(ptcls[idx].rs));
}

//---
template<class Model, class Controler, class Evaluator, class TmpSwarm>
__global__
void initialize_particles_kernel(Swarm sw, bool first_run,
                                 EDTMap map, Trace last_tr, Evaluator eva, Model m, Controler ctrl, TmpSwarm tsw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (first_run || idx != sw.ptcl_size-1)
  {
    m.initialize_a_particle(sw.ptcls[idx]);
  }
  sw.ptcls[idx].best_cost = evaluate_trajectory<Model,Controler,Evaluator,TmpSwarm>(sw.ptcls[idx].best_loc, map, last_tr,eva,m, ctrl, tsw.ptcls[idx].best_loc);

}

//---
template<class Model, class Controler, class Evaluator, class TmpSwarm>
__global__
void iterate_particles_kernel(Swarm sw, float weight,
                              EDTMap map, Trace last_tr, Evaluator eva, Model m, Controler ctrl, TmpSwarm tsw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (idx == sw.ptcl_size-1)
    return;

  float r1 = rand_float_gen(&(sw.ptcls[idx].rs),0,1);
  float r2 = rand_float_gen(&(sw.ptcls[idx].rs),0,1);

  sw.ptcls[idx].ptcl_vel =
      sw.ptcls[idx].ptcl_vel*weight -
      (sw.ptcls[idx].curr_loc - sw.ptcls[idx].best_loc)*r1 -
      (sw.ptcls[idx].curr_loc - sw.ptcls[sw.ptcl_size-1].curr_loc)*r2;

  m.bound_ptcl_velocity(sw.ptcls[idx]);

  sw.ptcls[idx].curr_loc = sw.ptcls[idx].curr_loc + sw.ptcls[idx].ptcl_vel;
  m.bound_ptcl_location(sw.ptcls[idx]);

  float cost = evaluate_trajectory<Model,Controler,Evaluator,TmpSwarm>(sw.ptcls[idx].curr_loc, map, last_tr,eva,m,ctrl,tsw.ptcls[idx].curr_loc);

  if (cost < sw.ptcls[idx].best_cost)
  {
    sw.ptcls[idx].best_cost = cost;
    sw.ptcls[idx].best_loc = sw.ptcls[idx].curr_loc;
  }
}

//---------
template<class TmpSwarm>
__global__
void copy_best_value_kernel(Particle *ptcls, float* best_values, TmpSwarm tsw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  best_values[idx] = ptcls[idx].best_cost;
}

//---------
template<class TmpSwarm>
void setup_random_states(const Swarm &sw, const TmpSwarm &tsw)
{
  setup_random_states_kernel<<<1,sw.ptcl_size>>>(sw.ptcls,tsw);
}

//---------
template<class Model, class Controler, class Evaluator, class TmpSwarm>
void initialize_particles(const Swarm &sw, bool first_run,
                          const EDTMap &map, const Trace &last_tr, const Evaluator &eva, const Model &m, const Controler &ctrl, const TmpSwarm &tsw)
{
  initialize_particles_kernel<Model,Controler,Evaluator,TmpSwarm><<<1,sw.ptcl_size>>>(sw,first_run,map, last_tr,eva,m,ctrl,tsw);
}

//---------
template<class Model, class Controler, class Evaluator, class TmpSwarm>
void iterate_particles(const Swarm &sw, float weight,
                       const EDTMap &map, const Trace &last_tr, const Evaluator &eva, const Model &m, const Controler &ctrl, const TmpSwarm &tsw)
{
  iterate_particles_kernel<Model,Controler,Evaluator,TmpSwarm><<<1,sw.ptcl_size>>>(sw,weight,map,last_tr,eva,m,ctrl,tsw);
}

//---------
template<class TmpSwarm>
void copy_best_values(const Swarm &sw, float *best_values, const TmpSwarm &tsw)
{
  copy_best_value_kernel<<<1,sw.ptcl_size>>>(sw.ptcls,best_values,tsw);
}

//float evaluate_trajectory_wrapper(const UAVModel::State &s0, const Trace &tr, VoidPtrCarrier ptr_car,const UniformBinCarrier &ubc,
//               const EDTMap &map, const Trace &last_tr)
//{
//  return 0;
////  return evaluate_trajectory(s0, goal, tr, ptr_car,ubc,
////                             map, last_tr);
//}

}

template void PSO::initialize_particles<UAV::UAVModel, UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(const Swarm &sw, bool first_run,
                                                                       const EDTMap &map, const Trace &last_tr, const UAV::SingleTargetEvaluator &eva, const UAV::UAVModel &m, const UAV::UAVDPControl &ctrl , const  UAV::UAVSwarm<1> &tsw);


template void PSO::iterate_particles<UAV::UAVModel,  UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(const Swarm &sw, float weight,
                                                                    const EDTMap &map, const Trace &last_tr, const UAV::SingleTargetEvaluator &eva, const UAV::UAVModel &m,const UAV::UAVDPControl &ctrl , const  UAV::UAVSwarm<1> &tsw);

template float PSO::evaluate_trajectory<UAV::UAVModel,  UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(const Trace &tr,
                          const EDTMap &map, const Trace &last_tr, const UAV::SingleTargetEvaluator &eva, UAV::UAVModel &m,const UAV::UAVDPControl &ctrl, const UAV::UAVSwarm<1>::Trace &ttr);

template void PSO::setup_random_states< UAV::UAVSwarm<1> >(const Swarm &sw, const UAV::UAVSwarm<1> &tsw);

template void PSO::copy_best_values< UAV::UAVSwarm<1> >(const Swarm &sw, float *best_values, const UAV::UAVSwarm<1> &tsw);
