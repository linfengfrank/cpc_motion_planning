#include "hip/hip_runtime.h"
#include <cpc_motion_planning/pso/pso_kernels.cuh>
#include <cuda_geometry/hip/hip_vector_types.h>
#include <cpc_motion_planning/uav/uav_single_target_evluator.h>
namespace PSO
{
//---
template<class Model, class Controler, class Evaluator, class Swarm>
__host__ __device__
float evaluate_trajectory(const EDTMap &map, const Evaluator &eva, Model &m, const Controler &ctrl, const Swarm &sw, const typename Swarm::Trace &ttr)
{
  typename Model::State s = m.get_ini_state();
  float cost = 0;
  float dt = PSO_SIM_DT;
  //float3 goal_p = goal.s.p;
  for (float t=0.0f; t<PSO_TOTAL_T; t+=dt)
  {
    int i = static_cast<int>(floor(t/sw.step_dt));
    if (i > sw.steps - 1)
      i = sw.steps - 1;

    float3 u = ctrl.dp_control(s, ttr[i]);
    m.model_forward(s,u,dt);

    cost += 0.1*sqrt(u.x*u.x + u.y*u.y + u.z*u.z);
    cost += eva.process_cost(s,map);

//    float3 ctr_pnt = tr[i];
//    float3 diff_tr = ctr_pnt - goal_p;

//    cost+= 0.05*sqrt(diff_tr.x*diff_tr.x + diff_tr.y*diff_tr.y + diff_tr.z*diff_tr.z);
  }
  cost += eva.final_cost(s,map);
//  Trace diff = tr - last_tr;
//  cost += sqrt(diff.square())/static_cast<float>(PSO_STEPS);
  return cost;
}

//---
template<class Swarm>
__global__
void setup_random_states_kernel(typename Swarm::Particle* tptcls)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(9876, idx, 0, &(tptcls[idx].rs));
}

//---
template<class Model, class Controler, class Evaluator, class Swarm>
__global__
void initialize_particles_kernel(bool first_run,
                                 EDTMap map, Evaluator eva, Model m, Controler ctrl, Swarm sw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (first_run || idx != sw.ptcl_size-1)
  {
    sw.initialize_a_particle(m.get_ini_state(),sw.ptcls[idx]);
  }
  sw.ptcls[idx].best_cost = evaluate_trajectory<Model,Controler,Evaluator,Swarm>(map,eva,m, ctrl, sw, sw.ptcls[idx].best_loc);
}

//---
template<class Model, class Controler, class Evaluator, class Swarm>
__global__
void iterate_particles_kernel(float weight,
                              EDTMap map, Evaluator eva, Model m, Controler ctrl, Swarm sw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (idx == sw.ptcl_size-1)
    return;

  float r1 = sw.rand_float_gen(&(sw.ptcls[idx].rs),0,1);
  float r2 = sw.rand_float_gen(&(sw.ptcls[idx].rs),0,1);

  sw.ptcls[idx].ptcl_vel =
      sw.ptcls[idx].ptcl_vel*weight -
      (sw.ptcls[idx].curr_loc - sw.ptcls[idx].best_loc)*r1 -
      (sw.ptcls[idx].curr_loc - sw.ptcls[sw.ptcl_size-1].curr_loc)*r2;

  sw.bound_ptcl_velocity(sw.ptcls[idx]);

  sw.ptcls[idx].curr_loc = sw.ptcls[idx].curr_loc + sw.ptcls[idx].ptcl_vel;

  sw.bound_ptcl_location(m.get_ini_state(), sw.ptcls[idx]);

  float cost = evaluate_trajectory<Model,Controler,Evaluator,Swarm>(map,eva,m,ctrl,sw,sw.ptcls[idx].curr_loc);

  if (cost < sw.ptcls[idx].best_cost)
  {
    sw.ptcls[idx].best_cost = cost;
    sw.ptcls[idx].best_loc = sw.ptcls[idx].curr_loc;
  }
}

//---------
template<class Swarm>
__global__
void copy_best_value_kernel(float* best_values, Swarm sw)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  best_values[idx] = sw.ptcls[idx].best_cost;
}

//---------
template<class Swarm>
void setup_random_states(const Swarm &sw)
{
  setup_random_states_kernel<Swarm><<<1,sw.ptcl_size>>>(sw.ptcls);
}

//---------
template<class Model, class Controler, class Evaluator, class Swarm>
void initialize_particles(bool first_run,
                          const EDTMap &map,const Evaluator &eva, const Model &m, const Controler &ctrl, const Swarm &sw)
{
  initialize_particles_kernel<Model,Controler,Evaluator,Swarm><<<1,sw.ptcl_size>>>(first_run,map,eva,m,ctrl,sw);
}

//---------
template<class Model, class Controler, class Evaluator, class Swarm>
void iterate_particles(float weight,
                       const EDTMap &map, const Evaluator &eva, const Model &m, const Controler &ctrl, const Swarm &sw)
{
  iterate_particles_kernel<Model,Controler,Evaluator,Swarm><<<1,sw.ptcl_size>>>(weight,map,eva,m,ctrl,sw);
}

//---------
template<class Swarm>
void copy_best_values(float *best_values, const Swarm &sw)
{
  copy_best_value_kernel<Swarm><<<1,sw.ptcl_size>>>(best_values,sw);
}

//float evaluate_trajectory_wrapper(const UAVModel::State &s0, const Trace &tr, VoidPtrCarrier ptr_car,const UniformBinCarrier &ubc,
//               const EDTMap &map, const Trace &last_tr)
//{
//  return 0;
////  return evaluate_trajectory(s0, goal, tr, ptr_car,ubc,
////                             map, last_tr);
//}

}

template void PSO::initialize_particles<UAV::UAVModel, UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(bool first_run,
                                                                       const EDTMap &map, const UAV::SingleTargetEvaluator &eva, const UAV::UAVModel &m, const UAV::UAVDPControl &ctrl , const  UAV::UAVSwarm<1> &sw);


template void PSO::iterate_particles<UAV::UAVModel,  UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(float weight,
                                                                    const EDTMap &map,const UAV::SingleTargetEvaluator &eva, const UAV::UAVModel &m,const UAV::UAVDPControl &ctrl , const  UAV::UAVSwarm<1> &sw);

template float PSO::evaluate_trajectory<UAV::UAVModel,  UAV::UAVDPControl, UAV::SingleTargetEvaluator, UAV::UAVSwarm<1> >(
                          const EDTMap &map, const UAV::SingleTargetEvaluator &eva, UAV::UAVModel &m,const UAV::UAVDPControl &ctrl, const UAV::UAVSwarm<1> &sw, const UAV::UAVSwarm<1>::Trace &ttr);

template void PSO::setup_random_states< UAV::UAVSwarm<1> >(const UAV::UAVSwarm<1> &sw);

template void PSO::copy_best_values< UAV::UAVSwarm<1> >(float *best_values, const UAV::UAVSwarm<1> &sw);
