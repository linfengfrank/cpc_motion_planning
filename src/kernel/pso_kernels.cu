#include "hip/hip_runtime.h"
#include <cpc_motion_planning/pso/pso_kernels.cuh>

namespace PSO
{
//---
template<int N>
__host__ __device__
float evaluate_trajectory(const State &s0, const State &goal, const Trace &tr, VoidPtrCarrier<N> ptr_car)
{
  State s = s0;
  float cost = 0;
  float dt = 0.05f;
  for (float t=0.0f; t<PSO_TOTAL_T; t+=dt)
  {
    int i = static_cast<int>(floor(t/PSO_STEP_DT));
    if (i > PSO_STEPS - 1)
      i = PSO_STEPS - 1;

    float3 u = dp_control<N>(s, tr[i], ptr_car);
    model_forward(s,u,dt);
    cost += process_cost(s,goal);
  }
  cost += final_cost(s,goal);
  return cost;
}

//---
__global__
void setup_random_states_kernel(Particle *ptcls)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(9876, idx, 0, &(ptcls[idx].rs));
}

//---
template <int N>
__global__
void initialize_particles_kernel(Particle *ptcls, int ptcl_size, bool first_run,
                                 State s0, State goal, VoidPtrCarrier<N> ptr_car)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  //if (first_run || idx != ptcl_size-1)
  //{
  initialize_a_particle(s0, ptcls[idx]);
  ptcls[idx].best_cost = evaluate_trajectory(s0, goal, ptcls[idx].best_loc, ptr_car);
  //}
}

//---
template <int N>
__global__
void iterate_particles_kernel(Particle *ptcls, int ptcl_size, float weight,
                              State s0, State goal, VoidPtrCarrier<N> ptr_car)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (idx == ptcl_size-1)
    return;

  float r1 = rand_float_gen(&(ptcls[idx].rs),0,1);
  float r2 = rand_float_gen(&(ptcls[idx].rs),0,1);

  ptcls[idx].ptcl_vel =
      ptcls[idx].ptcl_vel*weight -
      (ptcls[idx].curr_loc - ptcls[idx].best_loc)*r1 -
      (ptcls[idx].curr_loc - ptcls[ptcl_size-1].curr_loc)*r2;

  bound_ptcl_velocity(ptcls[idx]);

  ptcls[idx].curr_loc = ptcls[idx].curr_loc + ptcls[idx].ptcl_vel;
  bound_ptcl_location(ptcls[idx], s0);

  float cost = evaluate_trajectory(s0, goal, ptcls[idx].curr_loc, ptr_car);

  if (cost < ptcls[idx].best_cost)
  {
    ptcls[idx].best_cost = cost;
    ptcls[idx].best_loc = ptcls[idx].curr_loc;
  }
}

//---------
__global__
void copy_best_value_kernel(Particle *ptcls, float* best_values)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  best_values[idx] = ptcls[idx].best_cost;
}

//---------
void setup_random_states(Particle *ptcls, int size)
{
  setup_random_states_kernel<<<1,size>>>(ptcls);
}

//---------
template<int N>
void initialize_particles(Particle *ptcls, int ptcls_size, bool first_run,
                          const State &s, const State &goal,VoidPtrCarrier<N> ptr_car)
{
  initialize_particles_kernel<N><<<1,ptcls_size>>>(ptcls,ptcls_size,first_run,s,goal,ptr_car);
}

//---------
template<int N>
void iterate_particles(Particle *ptcls, int ptcls_size, float weight,
                       const State &s, const State &goal,VoidPtrCarrier<N> ptr_car)
{
  iterate_particles_kernel<N><<<1,ptcls_size>>>(ptcls,ptcls_size,weight,s,goal,ptr_car);
}

//---------
void copy_best_values(Particle *ptcls, int ptcls_size, float *best_values)
{
  copy_best_value_kernel<<<1,ptcls_size>>>(ptcls,best_values);
}
}

template void PSO::initialize_particles<5>(Particle *ptcls, int ptcls_size, bool first_run,
                          const State &s, const State &goal,VoidPtrCarrier<5> ptr_car);

template void PSO::iterate_particles<5>(Particle *ptcls, int ptcls_size, float weight,
                       const State &s, const State &goal,VoidPtrCarrier<5> ptr_car);
