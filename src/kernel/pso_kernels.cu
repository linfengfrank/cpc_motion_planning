#include "hip/hip_runtime.h"
#include <cpc_motion_planning/pso/pso_kernels.cuh>
#include <cuda_geometry/hip/hip_vector_types.h>

namespace PSO
{
//---
__host__ __device__
float evaluate_trajectory(const State &s0, const Target &goal, const Trace &tr, VoidPtrCarrier ptr_car,const UniformBinCarrier &ubc,
                          const EDTMap &map, const Trace &last_tr)
{
  State s = s0;
  float cost = 0;
  float dt = PSO_SIM_DT;
  float3 goal_p = goal.s.p;
  for (float t=0.0f; t<PSO_TOTAL_T; t+=dt)
  {
    int i = static_cast<int>(floor(t/PSO_STEP_DT));
    if (i > PSO_STEPS - 1)
      i = PSO_STEPS - 1;

    float3 u = dp_control(s, tr[i], ptr_car, ubc);
    model_forward(s,u,dt);

    cost += 0.1*sqrt(u.x*u.x + u.y*u.y + u.z*u.z);
    cost += process_cost(s,goal,map);

    float3 ctr_pnt = tr[i];
    float3 diff_tr = ctr_pnt - goal_p;

    cost+= 0.05*sqrt(diff_tr.x*diff_tr.x + diff_tr.y*diff_tr.y + diff_tr.z*diff_tr.z);
  }
  cost += final_cost(s,goal,map);
//  Trace diff = tr - last_tr;
//  cost += sqrt(diff.square())/static_cast<float>(PSO_STEPS);
  return cost;
}

//---
__global__
void setup_random_states_kernel(Particle *ptcls)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(9876, idx, 0, &(ptcls[idx].rs));
}

//---
__global__
void initialize_particles_kernel(Swarm sw, bool first_run,
                                 State s0, Target goal, VoidPtrCarrier ptr_car, UniformBinCarrier ubc,
                                 EDTMap map, Trace last_tr)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (first_run || idx != sw.ptcl_size-1)
  {
    initialize_a_particle(s0, sw.ptcls[idx]);
  }
  sw.ptcls[idx].best_cost = evaluate_trajectory(s0, goal, sw.ptcls[idx].best_loc, ptr_car, ubc, map, last_tr);

}

//---
__global__
void iterate_particles_kernel(Swarm sw, float weight,
                              State s0, Target goal, VoidPtrCarrier ptr_car,  UniformBinCarrier ubc,
                              EDTMap map, Trace last_tr)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (idx == sw.ptcl_size-1)
    return;

  float r1 = rand_float_gen(&(sw.ptcls[idx].rs),0,1);
  float r2 = rand_float_gen(&(sw.ptcls[idx].rs),0,1);

  sw.ptcls[idx].ptcl_vel =
      sw.ptcls[idx].ptcl_vel*weight -
      (sw.ptcls[idx].curr_loc - sw.ptcls[idx].best_loc)*r1 -
      (sw.ptcls[idx].curr_loc - sw.ptcls[sw.ptcl_size-1].curr_loc)*r2;

  bound_ptcl_velocity(sw.ptcls[idx]);

  sw.ptcls[idx].curr_loc = sw.ptcls[idx].curr_loc + sw.ptcls[idx].ptcl_vel;
  bound_ptcl_location(sw.ptcls[idx], s0);

  float cost = evaluate_trajectory(s0, goal, sw.ptcls[idx].curr_loc, ptr_car, ubc, map, last_tr);

  if (cost < sw.ptcls[idx].best_cost)
  {
    sw.ptcls[idx].best_cost = cost;
    sw.ptcls[idx].best_loc = sw.ptcls[idx].curr_loc;
  }
}

//---------
__global__
void copy_best_value_kernel(Particle *ptcls, float* best_values)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  best_values[idx] = ptcls[idx].best_cost;
}

//---------
void setup_random_states(const Swarm &sw)
{
  setup_random_states_kernel<<<1,sw.ptcl_size>>>(sw.ptcls);
}

//---------
void initialize_particles(const Swarm &sw, bool first_run,
                          const State &s, const Target &goal,VoidPtrCarrier ptr_car, const  UniformBinCarrier &ubc,
                          const EDTMap &map, const Trace &last_tr)
{
  initialize_particles_kernel<<<1,sw.ptcl_size>>>(sw,first_run,s,goal,ptr_car,ubc, map, last_tr);
}

//---------
void iterate_particles(const Swarm &sw, float weight,
                       const State &s, const Target &goal,VoidPtrCarrier ptr_car, const  UniformBinCarrier &ubc,
                       const EDTMap &map, const Trace &last_tr)
{
  iterate_particles_kernel<<<1,sw.ptcl_size>>>(sw,weight,s,goal,ptr_car,ubc,map,last_tr);
}

//---------
void copy_best_values(const Swarm &sw, float *best_values)
{
  copy_best_value_kernel<<<1,sw.ptcl_size>>>(sw.ptcls,best_values);
}

float evaluate_trajectory_wrapper(const State &s0, const Target &goal, const Trace &tr, VoidPtrCarrier ptr_car,const UniformBinCarrier &ubc,
               const EDTMap &map, const Trace &last_tr)
{
  return evaluate_trajectory(s0, goal, tr, ptr_car,ubc,
                             map, last_tr);
}

}

//template void PSO::initialize_particles<5>(const Swarm &sw, bool first_run,
//const State &s, const State &goal,VoidPtrCarrier<5> ptr_car, const  UniformBinCarrier &ubc,
//const EDTMap &map, const Trace &last_tr);

//template void PSO::iterate_particles<5>(const Swarm &sw, float weight,
//const State &s, const State &goal,VoidPtrCarrier<5> ptr_car, const  UniformBinCarrier &ubc,
//const EDTMap &map, const Trace &last_tr);

//template float PSO::evaluate_trajectory_wrapper<5>(const State &s0, const State &goal, const Trace &tr, VoidPtrCarrier<5> ptr_car,const UniformBinCarrier &ubc,
//const EDTMap &map, const Trace &last_tr);
